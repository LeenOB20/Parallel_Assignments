
#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void matrixMultiplication(float* a, float* b, float* c, int M, int N) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];
    float sum = 0.0;

    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        if (row < M && t * TILE_SIZE + tx < N) {
            tileA[ty][tx] = a[row * N + t * TILE_SIZE + tx];
        } else {
            tileA[ty][tx] = 0.0;
        }

        if (col < N && t * TILE_SIZE + ty < N) {
            tileB[ty][tx] = b[(t * TILE_SIZE + ty) * N + col];
        } else {
            tileB[ty][tx] = 0.0;
        }

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += tileA[ty][k] * tileB[k][tx];
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        c[row * N + col] = sum;
    }
}

int main() {
    int M = 1000;
    int N = 500;
    float* a, * b, * c;
    float* d_a, * d_b, * d_c;
    int size_a = M * N * sizeof(float);
    int size_b = N * N * sizeof(float);
    int size_c = M * N * sizeof(float);

    a = (float*)malloc(size_a);
    b = (float*)malloc(size_b);
    c = (float*)malloc(size_c);

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            a[i * N + j] = i + j;
        }
    }

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            b[i * N + j] = i - j;
        }
    }

    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);

    dim3 dimGrid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE, 1);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matrixMultiplication<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, M, N);

    hipEventRecord(stop);
    hipDeviceSynchronize();

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);

    

    printf("Execution time: %.2f ms\n", milliseconds);

    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

